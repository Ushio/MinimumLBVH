#include "hip/hip_runtime.h"
#include "minimum_lbvh.h"
#include "hip/hip_vector_types.h"

using namespace minimum_lbvh;

template <class T, class F>
__device__ T ReduceBlock(T val, T* smem, int blockDim, F f)
{
	smem[threadIdx.x] = val;
	__syncthreads();
	for (int i = 1; i < blockDim; i *= 2)
	{
		if (threadIdx.x < (threadIdx.x ^ i))
			smem[threadIdx.x] = f(smem[threadIdx.x], smem[threadIdx.x ^ i]);
		__syncthreads();
	}
	return smem[0];
}


// https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda/72461459#72461459
__device__ float atomicMinFloat(float* addr, float value) {
	float old;
	old = !signbit(value) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

__device__ float atomicMaxFloat(float* addr, float value) {
	float old;
	old = !signbit(value) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
		__uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

extern "C" __global__ void getSceneAABB(AABB* sceneAABB, const Triangle* triangles, int nTriangles)
{
	__shared__ float3 s_mem[256];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	AABB aabb = AABB::empty();
	if (idx < nTriangles)
	{
		Triangle tri = triangles[idx];
		for (int i = 0; i < 3; i++)
		{
			aabb.extend(tri.vs[i]);
		}
	}

	aabb.lower = ReduceBlock(aabb.lower, s_mem, blockDim.x, [](float3 a, float3 b) { return fminf(a, b); });
	aabb.upper = ReduceBlock(aabb.upper, s_mem, blockDim.x, [](float3 a, float3 b) { return fmaxf(a, b); });

	if (threadIdx.x == 0)
	{
		atomicMinFloat(&sceneAABB->lower.x, aabb.lower.x);
		atomicMinFloat(&sceneAABB->lower.y, aabb.lower.y);
		atomicMinFloat(&sceneAABB->lower.z, aabb.lower.z);
		atomicMaxFloat(&sceneAABB->upper.x, aabb.upper.x);
		atomicMaxFloat(&sceneAABB->upper.y, aabb.upper.y);
		atomicMaxFloat(&sceneAABB->upper.z, aabb.upper.z);
	}
}

extern "C" __global__ void buildMortons(IndexedMorton* indexedMortons, const Triangle *triangles, int nTriangles)
{
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//if (nTriangles <= idx)
	//{
	//	return;
	//}

	//Triangle tri = triangles[i];
	//float3 center = (tri.vs[0] + tri.vs[1] + tri.vs[2]) / 3.0f;
	//indexedMortons[i].morton = (uint32_t)(sceneAABB.encodeMortonCode(center) >> 31); // take higher 32bits out of 63bits
	//indexedMortons[i].index = i;
}